#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <tuple>
#include <hip/hip_runtime.h>

// Function to read the sparse matrix from a file
std::vector<std::tuple<int, int, float>> readSparseMatrix(const std::string& filename) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        throw std::runtime_error("Failed to open file");
    }

    std::vector<std::tuple<int, int, float>> matrix;
    int item1, item2;
    float value;
    while (file >> item1 >> item2 >> value) {
        matrix.push_back({item1, item2, value});
    }
    file.close();
    return matrix;
}

// Function to write the result to a file
void writeResultToFile(const std::vector<std::tuple<int, int, float>>& result, const std::string& filename) {
    std::ofstream file(filename);
    if (!file.is_open()) {
        throw std::runtime_error("Failed to open file");
    }

    for (const auto& [item1, item2, value] : result) {
        file << item1 << " " << item2 << " " << value << "\n";
    }
    file.close();
}

// CUDA kernel to perform matrix multiplication
__global__ void multiplyMatricesKernel(
    const std::tuple<int, int, float>* A,
    const std::tuple<int, int, float>* AT,
    std::tuple<int, int, float>* C,
    int numElements
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numElements) return;

    int item1 = std::get<0>(A[idx]);
    int item2 = std::get<1>(A[idx]);
    float value = std::get<2>(A[idx]);

    // Find matching elements in AT
    for (int j = 0; j < numElements; ++j) {
        if (std::get<0>(AT[j]) == item2) {
            int resultItem1 = item1;
            int resultItem2 = std::get<1>(AT[j]);
            float resultValue = value * std::get<2>(AT[j]);

            // Find if the result position already exists
            bool found = false;
            for (int k = 0; k < numElements; ++k) {
                if (std::get<0>(C[k]) == resultItem1 && std::get<1>(C[k]) == resultItem2) {
                    std::get<2>(C[k]) += resultValue;
                    found = true;
                    break;
                }
            }
            if (!found) {
                C[numElements + idx] = {resultItem1, resultItem2, resultValue};
            }
        }
    }
}

int main() {
    try {
        // Read the sparse matrix from file
        std::vector<std::tuple<int, int, float>> A = readSparseMatrix("matrix.txt");

        // Transpose matrix A
        std::vector<std::tuple<int, int, float>> AT;
        for (const auto& [item1, item2, value] : A) {
            AT.push_back({item2, item1, value});
        }

        // Allocate memory on GPU
        std::tuple<int, int, float>* d_A;
        std::tuple<int, int, float>* d_AT;
        std::tuple<int, int, float>* d_C;
        size_t size = A.size() * sizeof(std::tuple<int, int, float>);

        hipMalloc(&d_A, size);
        hipMalloc(&d_AT, size);
        hipMalloc(&d_C, size * 2); // Allocate extra space for results

        hipMemcpy(d_A, A.data(), size, hipMemcpyHostToDevice);
        hipMemcpy(d_AT, AT.data(), size, hipMemcpyHostToDevice);

        // Launch kernel
        int blockSize = 256;
        int numBlocks = (A.size() + blockSize - 1) / blockSize;
        multiplyMatricesKernel<<<numBlocks, blockSize>>>(d_A, d_AT, d_C, A.size());

        // Copy result back to host
        std::vector<std::tuple<int, int, float>> C(A.size() * 2);
        hipMemcpy(C.data(), d_C, size * 2, hipMemcpyDeviceToHost);

        // Write result to file
        writeResultToFile(C, "result.txt");

        // Free GPU memory
        hipFree(d_A);
        hipFree(d_AT);
        hipFree(d_C);

        std::cout << "Matrix multiplication completed successfully." << std::endl;

    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
    }

    return 0;
}