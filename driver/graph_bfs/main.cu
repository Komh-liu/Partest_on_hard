#include "hip/hip_runtime.h"
#include "graph_bfs.h"
#include <fstream>
#include <sstream>
#include <algorithm>
#include <chrono>

std::vector<int> loadFileToVector(const std::string& filename) {
    std::vector<int> result;
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        return result;
    }
    std::string line;
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        int num;
        while (iss >> num) {
            result.push_back(num);
        }
    }
    file.close();
    return result;
}

CUDAGraph loadCUDAGraphFromFile(const std::string& filename) {
    std::ifstream file(filename);
    CUDAGraph graph;
    std::vector<std::vector<int>> tempGraph;

    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        return {0, 0, nullptr, nullptr};
    }

    // 读取数据到临时邻接表
    int maxVertex = 0;
    std::string line;
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        int u, v;
        if (iss >> u >> v) {
            maxVertex = std::max({maxVertex, u, v});
            if (tempGraph.size() <= maxVertex) tempGraph.resize(maxVertex + 1);
            tempGraph[u].push_back(v);
            tempGraph[v].push_back(u);
        }
    }
    file.close();

    // 转换为CSR格式
    graph.numVertices = tempGraph.size();
    graph.offset = new int[graph.numVertices + 1];
    graph.offset[0] = 0;
    
    for (int i = 0; i < tempGraph.size(); ++i) {
        graph.offset[i+1] = graph.offset[i] + tempGraph[i].size();
    }
    
    graph.numEdges = graph.offset[tempGraph.size()];
    graph.edges = new int[graph.numEdges];
    
    int idx = 0;
    for (const auto& list : tempGraph) {
        for (int v : list) {
            graph.edges[idx++] = v;
        }
    }

    return graph;
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <input_file> <result_file>" << std::endl;
        return 1;
    }

    std::string input_file = argv[1];
    std::string result_file = argv[2];
    CUDAGraph graph = loadCUDAGraphFromFile(input_file);

    if (graph.numVertices > 0) {
        int bfs_start_vertex = 0; // 修改变量名以避免冲突
        std::cout << "BFS starting from vertex " << bfs_start_vertex << ":\n";
        std::vector<int> bfs_result;
        std::vector<int> result = loadFileToVector(result_file);
        // 时间测量部分
        auto time_start = std::chrono::high_resolution_clock::now(); // 修改变量名
        bfs(graph, bfs_start_vertex,bfs_result);
        auto time_end = std::chrono::high_resolution_clock::now(); // 修改变量名

        // 清理内存
        delete[] graph.offset;
        delete[] graph.edges;

        // 输出运行时间
        std::cout << "Time: " 
                  << std::chrono::duration_cast<std::chrono::milliseconds>(time_end - time_start).count()
                  << "ms\n";
        if(result == bfs_result)
            std::cout << "验证成功";
    }
    return 0;
}