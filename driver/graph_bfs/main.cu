#include "hip/hip_runtime.h"
#include "graph_bfs.h"
#include <fstream>
#include <sstream>
#include <algorithm>
#include <chrono>

std::vector<int> loadFileToVector(const std::string& filename) {
    std::vector<int> result;
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        return result;
    }
    std::string line;
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        int num;
        while (iss >> num) {
            result.push_back(num);
        }
    }
    file.close();
    return result;
}

CUDAGraph loadCUDAGraphFromFile(const std::string& filename) {
    std::ifstream file(filename);
    CUDAGraph graph;

    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        return {0, 0, nullptr, nullptr};
    }

    std::vector<int> tempEdges;
    std::vector<int> tempOffset;
    tempOffset.push_back(0);
    int maxVertex = 0;

    // First pass: Calculate maxVertex and the number of neighbors for each vertex
    std::string line;
    while (std::getline(file, line)) {
        std::replace(line.begin(), line.end(), ',', ' ');
        std::istringstream iss(line);
        int u, v;
        if (iss >> u >> v) {
            maxVertex = std::max({maxVertex, u, v});
            if (tempOffset.size() <= u + 1) tempOffset.resize(u + 2, 0);
            if (tempOffset.size() <= v + 1) tempOffset.resize(v + 2, 0);
            tempOffset[u + 1]++;
            tempOffset[v + 1]++; // Assuming undirected graph
        }
    }
    file.close();

    graph.numVertices = maxVertex + 1;
    graph.offset = new int[graph.numVertices + 1];
    std::partial_sum(tempOffset.begin(), tempOffset.end(), graph.offset);
    graph.numEdges = graph.offset[graph.numVertices];
    graph.edges = new int[graph.numEdges];

    std::vector<int> currentEdgeIndex(graph.numVertices, 0);

    // Second pass: Fill the edges array
    file.open(filename);
    if (!file.is_open()) {
        std::cerr << "无法重新打开文件: " << filename << std::endl;
        delete[] graph.offset;
        delete[] graph.edges;
        return {0, 0, nullptr, nullptr};
    }

    while (std::getline(file, line)) {
        std::replace(line.begin(), line.end(), ',', ' ');
        std::istringstream iss(line);
        int u, v;
        if (iss >> u >> v) {
            int indexU = graph.offset[u] + currentEdgeIndex[u];
            if (indexU < graph.numEdges) {
                graph.edges[indexU] = v;
                currentEdgeIndex[u]++;
            }

            int indexV = graph.offset[v] + currentEdgeIndex[v];
            if (indexV < graph.numEdges) {
                graph.edges[indexV] = u;
                currentEdgeIndex[v]++;
            }
        }
    }
    file.close();

    return graph;
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <input_file> <result_file>" << std::endl;
        return 1;
    }

    std::string input_file = argv[1];
    std::string result_file = argv[2];
    CUDAGraph graph = loadCUDAGraphFromFile(input_file);

    if (graph.numVertices > 0) {
        int bfs_start_vertex = 0; // 修改变量名以避免冲突
        std::cout << "BFS starting from vertex " << bfs_start_vertex << ":\n";
        std::vector<int> bfs_result;
        std::vector<int> result = loadFileToVector(result_file);
        // 时间测量部分
        auto time_start = std::chrono::high_resolution_clock::now(); // 修改变量名
        bfs(graph, bfs_start_vertex,bfs_result);
        auto time_end = std::chrono::high_resolution_clock::now(); // 修改变量名

        // 清理内存
        delete[] graph.offset;
        delete[] graph.edges;

        // 输出运行时间
        std::cout << "Time: " 
                  << std::chrono::duration_cast<std::chrono::milliseconds>(time_end - time_start).count()
                  << "ms\n";
        if(result == bfs_result)
            std::cout << "验证成功";
    }
    return 0;
}