#include "hip/hip_runtime.h"
#include "graph_bfs.h"
#include <fstream>
#include <sstream>
#include <algorithm>
#include <chrono>

CUDAGraph loadCUDAGraphFromFile(const std::string& filename) {
    std::ifstream file(filename);
    CUDAGraph graph;
    std::vector<std::vector<int>> tempGraph;

    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        return {0, 0, nullptr, nullptr};
    }

    // 读取数据到临时邻接表
    int maxVertex = 0;
    std::string line;
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        int u, v;
        if (iss >> u >> v) {
            maxVertex = std::max({maxVertex, u, v});
            if (tempGraph.size() <= maxVertex) tempGraph.resize(maxVertex + 1);
            tempGraph[u].push_back(v);
            tempGraph[v].push_back(u);
        }
    }
    file.close();

    // 转换为CSR格式
    graph.numVertices = tempGraph.size();
    graph.offset = new int[graph.numVertices + 1];
    graph.offset[0] = 0;
    
    for (int i = 0; i < tempGraph.size(); ++i) {
        graph.offset[i+1] = graph.offset[i] + tempGraph[i].size();
    }
    
    graph.numEdges = graph.offset[tempGraph.size()];
    graph.edges = new int[graph.numEdges];
    
    int idx = 0;
    for (const auto& list : tempGraph) {
        for (int v : list) {
            graph.edges[idx++] = v;
        }
    }

    return graph;
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <input_file> <result_file>" << std::endl;
        return 1;
    }

    std::string input_file = argv[1];
    std::string result_file = argv[2];
    CUDAGraph graph = loadCUDAGraphFromFile(input_file);

    if (graph.numVertices > 0) {
        int bfs_start_vertex = 0; // 修改变量名以避免冲突
        std::cout << "CUDA BFS starting from vertex " << bfs_start_vertex << ":\n";

        // 时间测量部分
        auto time_start = std::chrono::high_resolution_clock::now(); // 修改变量名
        bfs(graph, bfs_start_vertex);
        auto time_end = std::chrono::high_resolution_clock::now(); // 修改变量名

        // 清理内存
        delete[] graph.offset;
        delete[] graph.edges;

        // 输出运行时间
        std::cout << "Time: " 
                  << std::chrono::duration_cast<std::chrono::milliseconds>(time_end - time_start).count()
                  << "ms\n";
    }
    return 0;
}