#include "hip/hip_runtime.h"
#include "graph_bfs.h"
#include <fstream>
#include <fstream>
#include <sstream>
#include <algorithm>

CUDAGraph loadCUDAGraphFromFile(const std::string& filename) {
    std::ifstream file(filename);
    CUDAGraph graph;
    std::vector<std::vector<int>> tempGraph;

    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        return {0, 0, nullptr, nullptr};
    }

    // 读取数据到临时邻接表
    int maxVertex = 0;
    std::string line;
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        int u, v;
        if (iss >> u >> v) {
            maxVertex = std::max({maxVertex, u, v});
            if (tempGraph.size() <= maxVertex) tempGraph.resize(maxVertex + 1);
            tempGraph[u].push_back(v);
            tempGraph[v].push_back(u);
        }
    }
    file.close();

    // 转换为CSR格式
    graph.numVertices = tempGraph.size();
    graph.offset = new int[graph.numVertices + 1];
    graph.offset[0] = 0;
    
    for (int i = 0; i < tempGraph.size(); ++i) {
        graph.offset[i+1] = graph.offset[i] + tempGraph[i].size();
    }
    
    graph.numEdges = graph.offset[tempGraph.size()];
    graph.edges = new int[graph.numEdges];
    
    int idx = 0;
    for (const auto& list : tempGraph) {
        for (int v : list) {
            graph.edges[idx++] = v;
        }
    }

    return graph;
}

int main() {
    std::string filename = "data.txt";
    CUDAGraph graph = loadCUDAGraphFromFile(filename);

    if (graph.numVertices > 0) {
        int start = 0;
        std::cout << "CUDA BFS starting from vertex " << start << ":\n";
        bfs(graph, start);
        
        // 清理内存
        delete[] graph.offset;
        delete[] graph.edges;
    }

    return 0;
}