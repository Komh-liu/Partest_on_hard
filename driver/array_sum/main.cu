#include "array_sum.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>  // CUDA 运行时头文件
#include <chrono>

// 加载数组从文件
std::vector<int> load_array_from_file(const std::string& filename) {
    std::vector<int> arr;
    std::ifstream file(filename);
    std::string line;

    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        exit(1);
    }

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        int num;
        while (iss >> num) {
            arr.push_back(num);
        }
    }

    file.close();
    return arr;
}

int load_result_from_file(const std::string& filename) {
    int result;
    std::ifstream file(filename);
    std::string line;

    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        exit(1);
    }

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        int num;
        iss >> result;
    }

    file.close();
    return result;
}

// 主函数
int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <input_file> <result_file>" << std::endl;
        return 1;
    }

    std::string input_file = argv[1];
    std::string result_file = argv[2];

    // 从文件加载数组
    std::vector<int> arr = load_array_from_file(input_file);
    int result = load_result_from_file(result_file);

    // 调用统一接口函数
    auto start = std::chrono::high_resolution_clock::now();
    int sum = array_sum(arr);
    std::cout << "数组的和是: " << sum << std::endl; // 统一函数调用
    auto end = std::chrono::high_resolution_clock::now();

    // 输出耗时和验证结果
    std::cout << "Time: " 
              << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()
              << "ms\n";
    if(result == sum)
        std::cout << "验证成功" << std::endl;
    return 0;
}