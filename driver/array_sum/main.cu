#include "array_sum.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>  // CUDA 运行时头文件

// 加载数组从文件
std::vector<int> load_array_from_file(const std::string& filename) {
    std::vector<int> arr;
    std::ifstream file(filename);
    std::string line;

    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        exit(1);
    }

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        int num;
        while (iss >> num) {
            arr.push_back(num);
        }
    }

    file.close();
    return arr;
}

// 主函数
int main() {
    // 从文件加载数组
    std::vector<int> arr = load_array_from_file("/home/liu/Gitrepo/parwork/dataset/array_sum/data.txt");

    // 调用统一接口函数
    int sum = array_sum(arr);

    // 输出结果
    std::cout << "数组的和是: " << sum << std::endl;

    return 0;
}