#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

// BFS 内核函数：并行处理当前层的节点
__global__ void bfs_kernel(const int* d_offset, const int* d_edges,
                          int* d_visited, int* d_current, int* d_next,
                          int* current_size, int* next_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < *current_size) {
        int u = d_current[idx];
        int start = d_offset[u];
        int end = d_offset[u + 1];

        // 遍历当前节点的所有邻居
        for (int i = start; i < end; ++i) {
            int v = d_edges[i];
            // 原子操作标记访问，确保每个节点只被处理一次
            if (atomicExch(&d_visited[v], 1) == 0) {
                int pos = atomicAdd(next_size, 1);  // 原子操作分配位置
                d_next[pos] = v;
            }
        }
    }
}

// BFS 主函数
void bfs(const CUDAGraph& graph, int start) {
    // 分配设备内存
    int *d_offset, *d_edges, *d_visited;
    int *d_current, *d_next;
    int *d_current_size, *d_next_size;

    hipMalloc(&d_offset, (graph.numVertices + 1) * sizeof(int));
    hipMalloc(&d_edges, graph.numEdges * sizeof(int));
    hipMalloc(&d_visited, graph.numVertices * sizeof(int));
    hipMalloc(&d_current, graph.numVertices * sizeof(int));
    hipMalloc(&d_next, graph.numVertices * sizeof(int));
    hipMallocManaged(&d_current_size, sizeof(int));
    hipMallocManaged(&d_next_size, sizeof(int));

    // 拷贝图数据到设备
    hipMemcpy(d_offset, graph.offset, (graph.numVertices + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edges, graph.edges, graph.numEdges * sizeof(int), hipMemcpyHostToDevice);

    // 初始化 visited 数组和当前层
    hipMemset(d_visited, 0, graph.numVertices * sizeof(int));
    int one = 1;
    hipMemcpy(&d_visited[start], &one, sizeof(int), hipMemcpyHostToDevice);  // 标记起始节点
    *d_current_size = 1;
    *d_next_size = 0;
    hipMemcpy(d_current, &start, sizeof(int), hipMemcpyHostToDevice);

    // 打印起始节点
    std::cout << "BFS Order: " << start << " ";

    // 循环处理每一层
    while (*d_current_size > 0) {
        // 配置内核参数
        dim3 block(256);
        dim3 grid((*d_current_size + block.x - 1) / block.x);

        // 启动内核
        bfs_kernel<<<grid, block>>>(d_offset, d_edges, d_visited,
                                   d_current, d_next, d_current_size, d_next_size);
        hipDeviceSynchronize();

        // 交换 current 和 next，并重置 next_size
        std::swap(d_current, d_next);
        *d_current_size = *d_next_size;
        *d_next_size = 0;

        // 打印当前层的节点
        if (*d_current_size > 0) {
            int* current_layer = new int[*d_current_size];
            hipMemcpy(current_layer, d_current, *d_current_size * sizeof(int), hipMemcpyDeviceToHost);
            for (int i = 0; i < *d_current_size; ++i) {
                std::cout << current_layer[i] << " ";
            }
            delete[] current_layer;
        }
    }
    std::cout << std::endl;

    // 释放设备内存
    hipFree(d_offset);
    hipFree(d_edges);
    hipFree(d_visited);
    hipFree(d_current);
    hipFree(d_next);
    hipFree(d_current_size);
    hipFree(d_next_size);
}