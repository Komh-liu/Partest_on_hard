#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

using Matrix = std::vector<int>;

__global__ void matrixMultiplyKernel(const int* A, int* result, int N, int M) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < M; ++k) {
            sum += A[row * M + k] * A[col * M + k];  // 假设是 A * A^T
        }
        result[row * N + col] = sum;
    }
}

void matrix_multiply(const Matrix& A, int N, int M, Matrix& result) {
    // 检查矩阵是否为空
    if (A.empty() || N == 0 || M == 0) {
        std::cerr << "Error: Matrix is empty." << std::endl;
        return;
    }

    size_t sizeA = N * M * sizeof(int);
    size_t sizeResult = N * N * sizeof(int); // 结果矩阵的大小是 N x N

    int* d_A;
    int* d_result;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_result, sizeResult);

    // 检查CUDA错误
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA malloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    // 直接拷贝连续的一维主机数据到设备
    hipMemcpy(d_A, A.data(), sizeA, hipMemcpyHostToDevice);

    // 检查CUDA错误
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memcpy to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 调用内核函数进行矩阵乘法
    matrixMultiplyKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_result, N, M);

    // 检查CUDA错误
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    // 将结果从设备复制回主机
    result.resize(N * N);
    hipMemcpy(result.data(), d_result, sizeResult, hipMemcpyDeviceToHost);

    // 检查CUDA错误
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA memcpy from device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    hipFree(d_A);
    hipFree(d_result);
}